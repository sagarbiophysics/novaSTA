#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <algorithm>
#include "gpuRoutines.h"
#include "hip/hip_math_constants.h"
#include "quaternion.h"
#include "emFile.h"

texture<float, 3, hipReadModeElementType> texRef;      // 3D texture for reference
texture<float, 3, hipReadModeElementType> texSubtomo;  // 3D texture for subtomo
texture<float, 3, hipReadModeElementType> texWedge;    // 3D texture for wedge
texture<float, 3, hipReadModeElementType> texCCMask;    // 3D texture for ccmask
texture<float, 3, hipReadModeElementType> texMask;    // 3D texture for mask

using namespace std;

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) { getchar(); exit(code); }
    }
}

__global__ void gpu_sumArray(float *input, double *output, size_t elements)
{
    double localSum = 0;

    for (size_t i = threadIdx.x; i< elements; i += gridDim.x*blockDim.x)
    {
        if ((blockIdx.x*blockDim.x + i) >= elements)
            break;

        localSum += input[blockIdx.x*blockDim.x + i];
    }

    output[blockIdx.x*blockDim.x + threadIdx.x] = localSum;
}

__global__ void gpu_sumArraySquared(float* input, double *output, size_t elements)
{
    double localSum = 0;

    for (size_t i = threadIdx.x; i< elements; i += gridDim.x*blockDim.x)
    {
        if ((blockIdx.x*blockDim.x + i) >= elements)
            break;

        localSum = localSum + input[blockIdx.x*blockDim.x + i] * input[blockIdx.x*blockDim.x + i];
    }

    output[blockIdx.x*blockDim.x + threadIdx.x] = localSum;
}


__global__ void gpu_prepareNormValue(hipfftComplex* input, float* output, dim3 dim)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
    size_t k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if( i >= dim.x || j >= dim.y || k>= dim.z)
        return;
         
    size_t index = i + j*dim.x + k*dim.x*dim.y;
    float value = input[index].x*input[index].x + input[index].y*input[index].y;   
    
    if ( i==0 )
        output[index] = value;
    else
         output[index] = 2.0*value;
}

__global__ void gpu_computeNormFactor(hipfftComplex *input, double *output, size_t elements)
{
    double localSum = 0;

    for (size_t i = threadIdx.x; i< elements; i += gridDim.x*blockDim.x)
    {
        if ((blockIdx.x*blockDim.x + i) >= elements)
            break;

        localSum = localSum + (input[blockIdx.x*blockDim.x + i].x*input[blockIdx.x*blockDim.x + i].x) + (input[blockIdx.x*blockDim.x + i].y*input[blockIdx.x*blockDim.x + i].y);
    }

    output[blockIdx.x*blockDim.x + threadIdx.x] = localSum;
}

__device__  hipComplex my_expf(float scale, hipComplex value)
{
    hipComplex res;

    float t = expf(scale*value.x);
    sincosf(scale*value.y, &res.y, &res.x);

    res.x *= t;
    res.y *= t;
    
    return res;

}

__global__ void gpu_computeFLCC(float* ccf, float* numerator, float* intensitySubtomo, float* meanSubtomo, float maskSum, float sigmaRef, dim3 dim)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= dim.x || y >= dim.y || z >= dim.z)
        return;

    size_t shift = floor(dim.x / 2.0f) + 1;
    size_t volSize = dim.x*dim.y*dim.z;

    size_t inputIndex = volSize - 1 - (x + y*dim.x + z*dim.x*dim.y);
    size_t outputIndex = (x + shift) % dim.x + ((y + shift) % dim.y)*dim.x + ((z + shift) % dim.z)*dim.x*dim.y;
    double denominator = sqrt(intensitySubtomo[inputIndex] - meanSubtomo[inputIndex] * meanSubtomo[inputIndex] / maskSum)*sigmaRef;
    if (denominator != 0.0)
        ccf[outputIndex] = numerator[inputIndex] / denominator;
    else
        ccf[outputIndex] = -1.0f;
}

__global__ void gpu_generateShiftFilter(hipfftComplex* filter, int x, int y, int z, float3 shift)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int k = blockIdx.z * blockDim.z + threadIdx.z;

    float fftz = floor(z / 2.0f) + 1;

    float ox = floor(x / 2.0f);
    float oy = floor(y / 2.0f);
    float oz = floor(z / 2.0f);

    if ((i + (int)ox) % x >= fftz || j >= y || k >=z)
        return;

    shift.x /= (float)x;
    shift.y /= (float)y;
    shift.z /= (float)z;

    hipComplex c = { 0, 1 };

    double expValue = (i - ox)*shift.x + (j - oy)*shift.y + (k - oz)*shift.z;
  //  int voxelIndex = (k + (int)oz) % z + ((i + (int)ox) % x) * fftz + ((j + (int)oy) % y) * fftz*x;
    size_t voxelIndex = (i + (int)ox) % x + (j + (int)oy) % y * fftz + (k + (int)oz) % z* fftz*x;
    filter[voxelIndex] = my_expf(-2.0* HIP_PI_F *expValue, c);
}

__global__ void createWedgeMask(float* wedgeMask, int dim, float minAngle, float maxAngle)
{
    double minAngleRad = minAngle*HIP_PI_F / 180.0;
    double maxAngleRad = maxAngle*HIP_PI_F / 180.0;

    double tan_min = tan((-HIP_PI_F / 2.0) - minAngleRad);
    double tan_max = tan(HIP_PI_F / 2.0 - maxAngleRad);

    //wedgeMask.resize((size_t)dim*dim*dim);
//    fill(wedgeMask.begin(), wedgeMask.end(), 1.0f);

    float halfDim = floor(dim / 2.0);
    for (float z = -halfDim; z < -halfDim + dim; z++)
    {
        if (z == 0)
            continue;

        for (float x = -halfDim; x < -halfDim + dim; x++)
        {
            if ((tan_max > x / z) && (tan_min < x / z))
            {
                for (int y = 0; y < dim; y++)
                {
                    wedgeMask[(size_t)((x + halfDim) + y*dim + (z + halfDim)*dim*dim)] = 0.0f;
                }
            }
        }
    }
}

// Complex pointwise multiplication
__global__ void complexMultiplication(hipfftComplex* a, hipfftComplex* b, size_t elements)
{

    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= elements)
        return;

   // size_t i = ix + j*dim.x + k*dim.x*dim.y;
  
    float orValue = a[i].x;
    a[i].x = orValue * b[i].x - a[i].y * b[i].y;
    a[i].y = orValue * b[i].y + a[i].y * b[i].x;

}

// Complex pointwise multiplication
__global__ void gpu_complexMultiplication(hipfftComplex* a, hipfftComplex* b, hipfftComplex* c, size_t elements)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= elements)
        return;

    c[i].x = a[i].x * b[i].x - a[i].y * b[i].y;
    c[i].y = a[i].x * b[i].y + a[i].y * b[i].x;
}

// Complex pointwise conjugate multiplication with normalization
__global__ void gpu_conjugateMultiplication(hipfftComplex *a, hipfftComplex *b, float normFactor, size_t elements)
{
    //blockDim.x * gridDim.x;
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= elements)
        return;

    hipfftComplex cn;
    cn.x =  a[i].x * normFactor;
    cn.y = -a[i].y * normFactor;

    a[i].x = cn.x * b[i].x - cn.y * b[i].y;
    a[i].y = cn.x * b[i].y + cn.y * b[i].x;

}

// Complex pointwise multiplication
__global__ void gpu_applyCCMask(float* ref, float* mask, float* output, float volumeSize, dim3 dim, dim3 halfDim)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int k = blockIdx.z * blockDim.z + threadIdx.z;


    if (i >= dim.x || j >= dim.y || k >= dim.z)
        return;

    size_t input_index = i + j*dim.x + k*dim.x*dim.y;
    //size_t output_index = (k + halfDim.z) % dim.z + ((i + halfDim.x) % dim.x)*dim.z + ((j + halfDim.y) % dim.y)*dim.x*dim.z;
	size_t output_index = (i + halfDim.x) % dim.x + ((j + halfDim.y) % dim.y)*dim.z + ((k + halfDim.z) % dim.z)*dim.x*dim.z;
	
    output[output_index] = ref[input_index] * mask[output_index] / volumeSize;
}

// Complex pointwise masking
__global__ void complexMultiplicationDDD(hipfftComplex *a, float *mask, dim3 dim)
{
    //blockDim.x * gridDim.x;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= (dim.z * dim.y * dim.x))
        return;

    if (tid == 0)
    {
        a[tid].x = 0.0f;
        a[tid].y = 0.0f;
    }
    else
    {
        a[tid].x = a[tid].x * mask[tid];
        a[tid].y = a[tid].y * mask[tid];
    }
}

// Complex pointwise multiplication with a constant value
__global__ void gpu_complexMultiplication(hipfftComplex *a, float factor, size_t elements)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= elements)
        return;

    a[tid].x = a[tid].x * factor;
    a[tid].y = a[tid].y * factor;
}

// Complex pointwise multiplication with a constant value
__global__ void gpu_multiplication(float *a, float* b, size_t elements)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= elements)
        return;

    a[tid] = a[tid] * b[tid];
}

// Pointwise division
__global__ void division(float* a, float div, size_t elements)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= elements)
        return;

    a[tid] = a[tid] / div;
}

// Pointwise division
__global__ void gpu_subtract(float* a, float value, size_t elements)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= elements)
        return;
    
    if (a[tid]!=0.0f)
        a[tid] = a[tid] - value;
}

// Pointwise multiplication
__global__ void gpu_pointwiseMult(float* a, float* b, float* c, size_t elements)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= elements)
        return;

    c[tid] = a[tid] * b[tid];
}

// compute normalized square
__global__ void gpu_normalizedSquare(float *a, float factor, size_t elements)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= elements)
        return;

    a[tid] = a[tid] * a[tid] / factor;
}

// create complex conjugate
__global__ void gpu_complexConjugate(hipfftComplex* a, hipfftComplex* b, size_t elements)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= elements)
        return;

    a[tid].y = -a[tid].y;
    b[tid].y = -b[tid].y;
}

__device__  float binarizeValue(float value)
{
    float binValue;

    if (value >= 0.5f)
        binValue = 1.0f;
    else
        binValue = 0.0f;

    return binValue;
}


__device__ float interpolation(float* inputVolume, size_t inputIndex, float vx1, float vx2, float vy1, float vy2, float vz1, float vz2, size_t x, size_t y)
{
    float fb = inputVolume[inputIndex] + (inputVolume[inputIndex + 1] - inputVolume[inputIndex]) * vx2;
    float ft = inputVolume[inputIndex + x] * vx1 + inputVolume[inputIndex + x + 1] * vx2;
    float rb = inputVolume[inputIndex + x*y] * vx1 + inputVolume[inputIndex + x*y + 1] * vx2;
    float rt = inputVolume[inputIndex + x + x*y] * vx1 + inputVolume[inputIndex + x + x*y + 1] * vx2;

    float finalValue = (fb * vy1 + ft * vy2) * vz1 + (rb * vy1 + rt * vy2) * vz2;

    return finalValue;
}

__global__
void gpu_rotateVolume(float* volume, float* output, float* mask, float*maskOutput, float* ccMask, float*ccMaskOutput, size_t x, size_t y, size_t z, float4 quat, unsigned int rotationType)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= x || j >= y || k >= z)
        return;

    float cx = floor((float)x / 2.0f);
    float cy = floor((float)y / 2.0f);
    float cz = floor((float)z / 2.0f);

    long sxy = x * y;

    float3 rm1, rm2, rm3;
    float qi = quat.x;
    float qj = quat.y;
    float qk = quat.z;
    float w = quat.w;
    rm1.x = 1.0f - 2.0f*(qj*qj + qk*qk);
    rm1.y = 2.0f*(qi*qj + w*qk);
    rm1.z = 2.0f*(qi*qk - w*qj);

    rm2.x = 2.0f*(qi*qj - w*qk);
    rm2.y = 1.0f - 2.0f*(qi*qi + qk*qk);
    rm2.z = 2.0f*(qj*qk + w*qi);

    rm3.x = 2.0f*(qi*qk + w*qj);
    rm3.y = 2.0f*(qj*qk - w*qi);
    rm3.z = 1.0f - 2.0f*(qi*qi + qj*qj);

    long pi = i - cx;
    long pj = j - cy;
    long pk = k - cz;

    size_t outputIndex = i + j * x + k * sxy;

    /* transformation of coordinates */
    float rx = cx + rm1.x * pi + rm1.y * pj + rm1.z * pk;
    float ry = cy + rm2.x * pi + rm2.y * pj + rm2.z * pk;
    float rz = cz + rm3.x * pi + rm3.y * pj + rm3.z * pk;

    
    float floorx = floor(rx);

    if (rx < 0 || floorx >= (x - 1))
    {
        if (rotationType != 1 && rotationType != 4)
        {
            output[outputIndex] = 0.0f;
            maskOutput[outputIndex] = 0.0f;
            ccMaskOutput[outputIndex] = 0.0f;
        }
        return;
    }
    
    float floory = floor(ry);
    if (ry < 0 || floory >= (y - 1))
    {
        if (rotationType != 1 && rotationType != 4)
        {
            output[outputIndex] = 0.0f;
            maskOutput[outputIndex] = 0.0f;
            ccMaskOutput[outputIndex] = 0.0f;
        }
        return;
    }
    
    float floorz = floor(rz);
    if (rz < 0 || floorz >= (z - 1))
    {
        if (rotationType != 1 && rotationType != 4)
        {
            output[outputIndex] = 0.0f;
            maskOutput[outputIndex] = 0.0f;
            ccMaskOutput[outputIndex] = 0.0f;
        }
        return;
    }

    /* Interpolation */

    float vx2 = rx - floorx;
    float vx1 = 1 - vx2;

    float vy2 = ry - floory;
    float vy1 = 1 - vy2;

    float vz2 = rz - floorz;
    float vz1 = 1 - vz2;

    /* the following section detects border pixels to avoid exceeding dimensions */
    size_t inputIndex = floorx + floory * x + floorz * sxy;

    if (rotationType == 1)
    {
        output[outputIndex] += interpolation(volume, inputIndex, vx1, vx2, vy1, vy2, vz1, vz2,x,y);
        maskOutput[outputIndex] += binarizeValue(interpolation(mask, inputIndex, vx1, vx2, vy1, vy2, vz1, vz2, x, y));
    }
	else if (rotationType == 4)
    {
        output[outputIndex] += interpolation(volume, inputIndex, vx1, vx2, vy1, vy2, vz1, vz2,x,y);
        maskOutput[outputIndex] += interpolation(mask, inputIndex, vx1, vx2, vy1, vy2, vz1, vz2, x, y);
    }
    else
    {
        output[outputIndex] = interpolation(volume, inputIndex, vx1, vx2, vy1, vy2, vz1, vz2, x, y);
        ccMaskOutput[outputIndex] = interpolation(ccMask, inputIndex, vx1, vx2, vy1, vy2, vz1, vz2, x, y);
    }

    if (rotationType == 3)
    {
        maskOutput[outputIndex] = binarizeValue(interpolation(mask, inputIndex, vx1, vx2, vy1, vy2, vz1, vz2, x, y));
    }
}


__global__
void gpu_rotateTexture(float* output, float* mask, float* ccmask, float4 quat, size_t x, size_t y, size_t z, unsigned int type)
{

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int k = blockIdx.z * blockDim.z + threadIdx.z;

    if(i>=x || j>=y || k>=z)
        return;

    float cx = floor((float)x / 2.0f);
    float cy = floor((float)y / 2.0f);
    float cz = floor((float)z / 2.0f);

    // Rotation matrix computation

    float3 rm1, rm2, rm3;
    float qi = quat.x;
    float qj = quat.y;
    float qk = quat.z;
    float w = quat.w;
    rm1.x = 1.0f - 2.0f*(qj*qj + qk*qk);
    rm1.y = 2.0f*(qi*qj + w*qk);
    rm1.z = 2.0f*(qi*qk - w*qj);

    rm2.x = 2.0f*(qi*qj - w*qk);
    rm2.y = 1.0f - 2.0f*(qi*qi + qk*qk);
    rm2.z = 2.0f*(qj*qk + w*qi);

    rm3.x = 2.0f*(qi*qk + w*qj);
    rm3.y = 2.0f*(qj*qk - w*qi);
    rm3.z = 1.0f - 2.0f*(qi*qi + qj*qj);


    long pi = i - cx;
    long pj = j - cy;
    long pk = k - cz;

    // transformation of coordinates 
    float rx = cx + rm1.x * pi + rm1.y * pj + rm1.z * pk + 0.5f;
    float ry = cy + rm2.x * pi + rm2.y * pj + rm2.z * pk + 0.5f;
    float rz = cz + rm3.x * pi + rm3.y * pj + rm3.z * pk + 0.5f;

    // read from 3D texture
    if (type == 1)
    {
        output[i + j*x + k*x*y] += tex3D(texSubtomo, rx, ry, rz);
        mask[i + j*x + k*x*y] += binarizeValue(tex3D(texWedge, rx, ry, rz));
    }
    else if (type == 4)
    {
        output[i + j*x + k*x*y] += tex3D(texSubtomo, rx, ry, rz);
        mask[i + j*x + k*x*y] += tex3D(texWedge, rx, ry, rz);
    }
    else
    {
        output[i + j*x + k*x*y] = tex3D(texRef, rx, ry, rz);
        ccmask[i + j*x + k*x*y] = tex3D(texCCMask, rx, ry, rz);
    }

    if (type == 3)
    {
        mask[i + j*x + k*x*y] = binarizeValue(tex3D(texMask, rx, ry, rz));
    }
}

void GPURoutines::fft2DR2C(float* input, float* output_real, float* output_img, size_t x, size_t y)
{
    size_t ty = floor(y / 2) + 1;

    size_t numbytes = x*y*sizeof(float);
    size_t numel = x*y;

    float* fft_in;
    hipMalloc((void**)&fft_in, sizeof(float)*numel);

    //hipMalloc((void**)&dev_in, numbytes);
    hipMemcpy(fft_in, input, numbytes, hipMemcpyHostToDevice);

    hipfftComplex* fft_out;
    hipMalloc((void**)&fft_out, sizeof(hipfftComplex)*x*ty);

    // CUFFT plan simple API
    hipfftHandle plan;
    hipfftPlan2d(&plan, x, y, HIPFFT_R2C);

    // Transform signal and kernel
    hipfftExecR2C(plan, fft_in, fft_out);

    // Copy device memory to host
    vector<float2> outcome(x*ty);

    hipMemcpy(&outcome[0], fft_out, x*ty*sizeof(float2), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < x*ty; i++)
    {
        output_real[i] = outcome[i].x;
        output_img[i] = outcome[i].y;
    }

}

void GPURoutines::fft2DC2R(float* input_real, float* input_img, float* output, size_t x, size_t y)
{
    size_t ty = floor(y / 2) + 1;
    vector<float2> complex_in(x*ty);

    for (size_t i = 0; i < x*ty; i++)
    {
        complex_in[i].x = input_real[i];
        complex_in[i].y = input_img[i];
    }

    hipfftComplex* fft_in;
    hipMalloc((void**)&fft_in, sizeof(hipfftComplex)*x*ty);

    hipMemcpy(fft_in, &complex_in[0], sizeof(hipfftComplex)*x*ty, hipMemcpyHostToDevice);

    hipfftHandle planBack;
    hipfftPlan2d(&planBack, x, y, HIPFFT_C2R);

    float* fft_out;
    hipMalloc((void**)&fft_out, sizeof(float)*x*y);

    // Transform signal and kernel
    hipfftExecC2R(planBack, fft_in, fft_out);

    hipMemcpy(&output[0], fft_out, x*y*sizeof(float), hipMemcpyDeviceToHost);
}



void GPURoutines::fft3DR2C(float* input, float* output_real, float* output_img, size_t x, size_t y, size_t z)
{
    size_t tz = floor(z / 2) + 1;

    size_t numbytes = x*y*z*sizeof(float);
    size_t numel = x*y*z;

    float* fft_in;
    hipMalloc((void**)&fft_in, sizeof(float)*numel);

    //hipMalloc((void**)&dev_in, numbytes);
    hipMemcpy(fft_in, input, numbytes, hipMemcpyHostToDevice);

    hipfftComplex* fft_out;
    hipMalloc((void**)&fft_out, sizeof(hipfftComplex)*x*y*tz);

    // CUFFT plan simple API
    hipfftHandle plan;
    hipfftPlan3d(&plan, x, y, z, HIPFFT_R2C);

    // Transform signal and kernel
    hipfftExecR2C(plan, fft_in, fft_out);

    // Copy device memory to host
    vector<float2> outcome(x*y*tz);

    hipMemcpy(&outcome[0], fft_out, x*y*tz*sizeof(float2), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < x*y*tz; i++)
    {
        output_real[i] = outcome[i].x;
        output_img[i] = outcome[i].y;
    }
}


void GPURoutines::fft3DC2R(float* input_real, float* input_img, float* output, size_t x, size_t y, size_t z)
{
    size_t tz = floor(z / 2) + 1;
    vector<float2> complex_in(x*y*tz);

    for (size_t i = 0; i < x*y*tz; i++)
    {
        complex_in[i].x = input_real[i];
        complex_in[i].y = input_img[i];
    }

    hipfftComplex* fft_in;
    hipMalloc((void**)&fft_in, sizeof(hipfftComplex)*x*y*tz);

    hipMemcpy(fft_in, &complex_in[0], sizeof(hipfftComplex)*x*y*tz, hipMemcpyHostToDevice);

    hipfftHandle planBack;
    hipfftPlan3d(&planBack, x, y, z, HIPFFT_C2R);

    float* fft_out;
    hipMalloc((void**)&fft_out, sizeof(float)*x*y*z);

    // Transform signal and kernel
    hipfftExecC2R(planBack, fft_in, fft_out);

    hipMemcpy(&output[0], fft_out, x*y*z*sizeof(float), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < x*y*z; i++)
    {
        output[i] = output[i] / (float)(x*y*z);
    }
}

void GPURoutines::fft3DC2C(float* input_real, float* input_img, float* output, size_t x, size_t y, size_t z)
{
    vector<float2> complex_in(x*y*z);

    for (size_t i = 0; i < x*y*z; i++)
    {
        complex_in[i].x = input_real[i];
        complex_in[i].y = input_img[i];
    }

    hipfftComplex* fft_in;
    hipMalloc((void**)&fft_in, sizeof(hipfftComplex)*x*y*z);

    hipMemcpy(fft_in, &complex_in[0], sizeof(hipfftComplex)*x*y*z, hipMemcpyHostToDevice);

    hipfftHandle planBack;
    hipfftPlan3d(&planBack, x, y, z, HIPFFT_C2C);

    hipfftComplex* fft_out;
    hipMalloc((void**)&fft_out, sizeof(hipfftComplex)*x*y*z);

    // Transform signal and kernel
    hipfftExecC2C(planBack, fft_in, fft_out, HIPFFT_BACKWARD);

    vector<float2> tempOut(x*y*z);
    hipMemcpy(&tempOut[0], fft_out, x*y*z*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < x*y*z; i++)
    {
        output[i] = tempOut[i].x/(float)(x*y*z);
    }

}

GPURoutines::GPURoutines(size_t dimx, size_t dimy, size_t dimz, bool useRoseman,int processID)
{
    volumeDim = dim3( dimx, dimy, dimz );
    volumeSize = dimx*dimy*dimz;

    fftz = floor((float)volumeDim.z / 2) + 1;
    fftSize = dimx*dimy*fftz;

    // Create 3D array
    volumeExtent = make_hipExtent(dimx, dimy, dimz);

    // Create a channel description
    channelDesc = hipCreateChannelDesc<float>();

    threadsPerBlock1D = (8*8*8);  // 576 threads
    numBlocks1D = ceil(volumeSize / (float)threadsPerBlock1D.x);

    threadsPerBlock = { 8, 8, 8 };  // 512 threads
    numBlocks = dim3(ceil(dimx / (float)threadsPerBlock.x), ceil(dimy / (float)threadsPerBlock.y), ceil(dimz / (float)threadsPerBlock.z));

    threadsPerBlockFFT = { 8, 8, 8 };  // 576 threads
    numBlocksFFT = dim3(ceil(dimx / (float)threadsPerBlockFFT.x), ceil(dimy / (float)threadsPerBlockFFT.y), ceil(dimz / (float)threadsPerBlockFFT.z));

    subtomoAllocated = false;

    useRosemanCC = useRoseman;

    if (useRosemanCC)
        rotationType = 3;
    else
        rotationType = 2;
        
    textureAllocated = false;
  	
	distributeDevices(processID);
}

void GPURoutines::distributeDevices(int processID)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    int deviceID=processID%deviceCount;
    hipSetDevice(deviceID);
  
    cout << "Process #" << processID << " got card #" << deviceID << endl;   
}

GPURoutines::~GPURoutines()
{
    hipFree(shiftFilter);
    
    if (subtomoAllocated)
        hipFree(fftSubtomoInNew);

    hipFree(fftSubtomoOut);

    hipFree(deviceSubtomoRot);
    hipFree(deviceWedgeRot);
    hipFree(deviceCCMaskRot);
    hipFree(deviceRefRot);
   
    hipfftDestroy(planR2C);
    hipfftDestroy(planC2R);

 //   hipFree(rrrRef);
 //   hipFree(rrrMask);
 //   hipFree(rrrCCMask);

    if (useRosemanCC)
    {
        hipFree(deviceMaskRot);
        hipFree(conjugateSquare);
        hipFree(intensitySubtomo);
        hipFree(meanSubtomo);
        hipFree(meanMasked);
        hipFree(numerator);
    }

    if(textureAllocated)
    {
        hipFreeArray(deviceRef);
        hipFreeArray(deviceCCMask);
        hipFreeArray(deviceMask);
    }
}


void GPURoutines::prepareReferenceTexture(vector<float>& reference, vector<float>& mask, vector<float>& ccMask, string filterModeName)
{
    hipTextureFilterMode filterMode;

    if (filterModeName == "linear")
        filterMode = hipFilterModeLinear;
    else
        filterMode = hipFilterModePoint;

    // Allocate memory on device
    hipMalloc3DArray(&deviceRef, &channelDesc, volumeExtent);
    hipMalloc3DArray(&deviceCCMask, &channelDesc, volumeExtent);
    hipMalloc3DArray(&deviceMask, &channelDesc, volumeExtent);
    
    textureAllocated = true;
    
    // --- Set texture parameters
    texRef.normalized = false;                          // access with normalized texture coordinates
    texRef.filterMode = filterMode;                     // trilinear or no interpolation
    texRef.addressMode[0] = hipAddressModeBorder;      // wrap texture coordinates
    texRef.addressMode[1] = hipAddressModeBorder;
    texRef.addressMode[2] = hipAddressModeBorder;

    texCCMask.normalized = false;                       // access with normalized texture coordinates
    texCCMask.filterMode = filterMode;                     // trilinear or no interpolation
    texCCMask.addressMode[0] = hipAddressModeBorder;      // wrap texture coordinates
    texCCMask.addressMode[1] = hipAddressModeBorder;
    texCCMask.addressMode[2] = hipAddressModeBorder;

    // Set copy parameters for 3D array
    hipMemcpy3DParms copyParamsReference = { 0 };
    copyParamsReference.dstArray = deviceRef;
    copyParamsReference.extent = volumeExtent;
    copyParamsReference.kind = hipMemcpyHostToDevice;
    copyParamsReference.srcPtr = make_hipPitchedPtr((void *)&reference[0], volumeExtent.width*sizeof(float), volumeExtent.width, volumeExtent.height);

    // Copy 3D array from host to device
    gpuErrchk(hipMemcpy3D(&copyParamsReference));

    // Bind array to 3D texture
    gpuErrchk(hipBindTextureToArray(texRef, deviceRef, channelDesc));

    // Allocate memory for rotated reference
    gpuErrchk(hipMalloc((void **)&deviceRefRot, volumeSize*sizeof(float)));
    
    // Set copy parameters for 3D array
    copyParamsReference.dstArray = deviceCCMask;
    copyParamsReference.srcPtr = make_hipPitchedPtr((void *)&ccMask[0], volumeExtent.width*sizeof(float), volumeExtent.width, volumeExtent.height);

    // Copy 3D array from host to device
    gpuErrchk(hipMemcpy3D(&copyParamsReference));

    // Bind array to 3D texture
    gpuErrchk(hipBindTextureToArray(texCCMask, deviceCCMask, channelDesc));
    gpuErrchk(hipMalloc((void **)&deviceCCMaskRot, volumeSize*sizeof(float)));

   
    if (useRosemanCC)
    {
        // Set copy parameters for 3D array
        copyParamsReference.dstArray = deviceMask;
        copyParamsReference.srcPtr = make_hipPitchedPtr((void *)&mask[0], volumeExtent.width*sizeof(float), volumeExtent.width, volumeExtent.height);

        // Copy 3D array from host to device
        gpuErrchk(hipMemcpy3D(&copyParamsReference));

        // Bind array to 3D texture
        gpuErrchk(hipBindTextureToArray(texMask, deviceMask, channelDesc));
        gpuErrchk(hipMalloc((void **)&deviceMaskRot, volumeSize*sizeof(float)));
    }

  //  gpuErrchk(hipMalloc((void **)&deviceMaskRot, volumeSize*sizeof(float)));

  //  gpuErrchk(hipMalloc((void **)&rrrRef, volumeSize*sizeof(float)));
  //  gpuErrchk(hipMemcpy(rrrRef, &reference[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));
  
  //  gpuErrchk(hipMalloc((void **)&rrrMask, volumeSize*sizeof(float)));
  //  gpuErrchk(hipMemcpy(rrrMask, &mask[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));

  //  gpuErrchk(hipMalloc((void **)&rrrCCMask, volumeSize*sizeof(float)));
   // gpuErrchk(hipMemcpy(rrrCCMask, &ccMask[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));

    // Just for now to prevent zero rotation
    //gpuErrchk(hipMemcpy(deviceRefRot, &reference[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(deviceCCMaskRot, &ccMask[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));
    //gpuErrchk(hipMemcpy(deviceMaskRot, &mask[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));
}

void GPURoutines::prepareSubtomoTexture(string filterModeName)
{
    hipTextureFilterMode filterMode;

    if (filterModeName == "linear")
        filterMode = hipFilterModeLinear;
    else
        filterMode = hipFilterModePoint;
    
    // Allocate memory on device
    hipMalloc3DArray(&deviceSubtomo, &channelDesc, volumeExtent);
    hipMalloc3DArray(&deviceWedge, &channelDesc, volumeExtent);


    // --- Set texture parameters
    texSubtomo.normalized = false;                      // access with normalized texture coordinates
    texSubtomo.filterMode = filterMode;                 // trilinear interpolation
    texSubtomo.addressMode[0] = hipAddressModeBorder;  // wrap texture coordinates
    texSubtomo.addressMode[1] = hipAddressModeBorder;
    texSubtomo.addressMode[2] = hipAddressModeBorder;

    // --- Set texture parameters
    texWedge.normalized = false;                      // access with normalized texture coordinates
    texWedge.filterMode = filterMode;                 // trilinear interpolation
    texWedge.addressMode[0] = hipAddressModeBorder;  // wrap texture coordinates
    texWedge.addressMode[1] = hipAddressModeBorder;
    texWedge.addressMode[2] = hipAddressModeBorder;

    // Set copy parameters for 3D array
    copyParamsSubtomo = { 0 };
    copyParamsSubtomo.dstArray = deviceSubtomo;
    copyParamsSubtomo.extent = volumeExtent;
    copyParamsSubtomo.kind = hipMemcpyDeviceToDevice;
    
    // Set copy parameters for 3D array
    copyParamsWedge = { 0 };
    copyParamsWedge.dstArray = deviceWedge;
    copyParamsWedge.extent = volumeExtent;
    copyParamsWedge.kind = hipMemcpyHostToDevice;
    
}

void GPURoutines::copyAndBindWedgeTexture(vector<float>& wedgeMask)
{
    //double minAngleRad = minAngle*HIP_PI_F / 180.0;
    //double maxAngleRad = maxAngle*HIP_PI_F / 180.0;

    //double tan_min = tan((-HIP_PI_F / 2.0) - minAngleRad);
    //double tan_max = tan(HIP_PI_F / 2.0 - maxAngleRad);

    //wedgeMask.resize((size_t)dim*dim*dim);
    //fill(wedgeMask.begin(), wedgeMask.end(), 1.0f);

    
    
    copyParamsWedge.srcPtr = make_hipPitchedPtr((void *)&wedgeMask[0], volumeExtent.width*sizeof(float), volumeExtent.width, volumeExtent.height);

    // Copy 3D array from host to device
    gpuErrchk(hipMemcpy3D(&copyParamsWedge));

    // Bind array to 3D texture
    gpuErrchk(hipBindTextureToArray(texWedge, deviceWedge, channelDesc));
}

void GPURoutines::rotateVectorIndices(vector<int>& ids, size_t vectorSize, int direction)
{
    size_t shiftStart;
    
    if (direction==-1)
        shiftStart = (size_t)ceil((float)vectorSize / 2.0f);
    else
        shiftStart = (size_t)floor((float)vectorSize / 2.0f);

    for (int i = 0; i < vectorSize; i++)
        ids.push_back(i);

    rotate(ids.begin(), ids.begin() + shiftStart, ids.end());
}

void GPURoutines::allocateDeviceMemory()
{
    // allocate memory for shift filter
    gpuErrchk(hipMalloc((void **)&shiftFilter, fftSize*sizeof(hipfftComplex)));
    gpuErrchk(hipMemset(shiftFilter, 0.0, fftSize*sizeof(hipfftComplex)));
   
  //  gpuErrchk(hipMalloc((void **)&deviceSubtomoRot, volumeSize*sizeof(float)));
    gpuErrchk(hipMalloc((void **)&deviceWedgeRot, volumeSize*sizeof(float)));

    gpuErrchk(hipMemset(deviceWedgeRot, 0.0f, volumeSize*sizeof(float)));
  //  gpuErrchk(hipMemset(deviceSubtomoRot, 0.0f, volumeSize*sizeof(float)));

    hipfftPlan3d(&planR2C, volumeDim.x, volumeDim.y, volumeDim.z, HIPFFT_R2C);
    hipfftPlan3d(&planC2R, volumeDim.x, volumeDim.y, volumeDim.z, HIPFFT_C2R);

 
    // allocate memory for subtomogram for FFT output
    gpuErrchk(hipMalloc((void **)&fftSubtomoOut, fftSize*sizeof(hipfftComplex)));

    gpuErrchk(hipMalloc((void **)&deviceSubtomoRot, volumeSize*sizeof(float)));
    gpuErrchk(hipMemset(deviceSubtomoRot, 0.0f, volumeSize*sizeof(float)));

    hipDeviceSynchronize();

    if (useRosemanCC)
    {
        gpuErrchk(hipMalloc((void **)&numerator, volumeSize*sizeof(float)));
        gpuErrchk(hipMalloc((void **)&meanSubtomo, volumeSize*sizeof(float)));
        gpuErrchk(hipMalloc((void **)&intensitySubtomo, volumeSize*sizeof(float)));
        gpuErrchk(hipMalloc((void **)&meanMasked, fftSize*sizeof(hipfftComplex)));
    }
}

void GPURoutines::shiftSubtomogram(vector<float>& subtomo, float shiftX, float shiftY, float shiftZ)
{
    float3 shift = { shiftX, shiftY, shiftZ };
   
    gpuErrchk(hipMalloc((void **)&fftSubtomoInNew, volumeSize*sizeof(float)));
    gpuErrchk(hipMemcpy(fftSubtomoInNew, &subtomo[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));
    subtomoAllocated = true;

    hipDeviceSynchronize();

    gpu_generateShiftFilter << <numBlocks, threadsPerBlock >> > (shiftFilter, volumeDim.x, volumeDim.y, volumeDim.z, shift);
   
    // Transform signal and kernel
    hipfftExecR2C(planR2C, fftSubtomoInNew, fftSubtomoOut);
    hipDeviceSynchronize();
    
    complexMultiplication << <numBlocks1D, threadsPerBlock1D >> >(fftSubtomoOut, shiftFilter, fftSize);

    hipDeviceSynchronize();
 
    // Transform signal and kernel
    hipfftExecC2R(planC2R, fftSubtomoOut, fftSubtomoInNew);
    hipDeviceSynchronize();

    division << <numBlocks1D, threadsPerBlock1D >> > (fftSubtomoInNew, volumeSize, volumeSize);

}

double GPURoutines::sumArray(vector<float>& data)
{
    float* deviceArray;
    hipMalloc((void**)&deviceArray, sizeof(float)*data.size());
    hipMemcpy(deviceArray, &data[0], sizeof(float)*data.size(), hipMemcpyHostToDevice);

    double sum = sumArray(deviceArray, data.size());

    hipFree(deviceArray);

    return sum;
}

double GPURoutines::computeNormFactor(vector<float2>& data)
{
    hipfftComplex* deviceArray;
    hipMalloc((void**)&deviceArray, sizeof(hipfftComplex)*data.size());
    hipMemcpy(deviceArray, &data[0], sizeof(hipfftComplex)*data.size(), hipMemcpyHostToDevice);

    double nf = computeNormFactor(deviceArray, data.size());

    hipFree(deviceArray);

    return nf;
}

double GPURoutines::sumArray(float* deviceArray, size_t arraySize, bool squared)
{
    unsigned int threads = 512;
    unsigned int blocks = 2;

    double* deviceOut;
    hipMalloc((void**)&deviceOut, sizeof(double)*threads*blocks);

    if (squared)
        gpu_sumArraySquared << <blocks, threads >> >(deviceArray, deviceOut, arraySize);
    else
        gpu_sumArray << <blocks, threads >> >(deviceArray, deviceOut, arraySize);

    vector<double> out(threads*blocks);
    hipMemcpy(&out[0], deviceOut, sizeof(double)*threads*blocks, hipMemcpyDeviceToHost);

    double globalSum = 0;
    for (size_t i = 0; i < threads*blocks; i++)
        globalSum += out[i];

    hipFree(deviceOut);
    return globalSum;
}

double GPURoutines::computeNormFactor(hipfftComplex* deviceArray, size_t arraySize)
{

    float* conjugateValues;
    hipMalloc((void**)&conjugateValues, sizeof(float)*fftSize);
    
    dim3 fftDim(fftz, volumeDim.y, volumeDim.z);    
    gpu_prepareNormValue<< <numBlocks, threadsPerBlock >> >(deviceArray,conjugateValues,fftDim);
    
    
    unsigned int threads = 512;
    unsigned int blocks = 1;

    double* deviceOut;
    hipMalloc((void**)&deviceOut, sizeof(double)*threads*blocks);

    gpu_sumArray << <blocks, threads >> >(conjugateValues, deviceOut, arraySize);

    vector<double> out(threads*blocks);
    hipMemcpy(&out[0], deviceOut, sizeof(double)*threads*blocks, hipMemcpyDeviceToHost);

    double globalSum = 0;
    for (size_t i = 0; i < threads*blocks; i++)
    {
        globalSum = globalSum + out[i];
    }
    
    hipFree(conjugateValues);
	hipFree(deviceOut);
    return globalSum;
}

void GPURoutines::computeConjugateOfSquare()
{
   // hipfftComplex* conjugateSquare;
    gpuErrchk(hipMalloc((void **)&conjugateSquare, fftSize*sizeof(hipfftComplex)));

    float* fft_out;
    hipMalloc((void**)&fft_out, sizeof(float)*volumeSize);

    hipfftHandle planBack;
    hipfftPlan3d(&planBack, volumeDim.x, volumeDim.y, volumeDim.z, HIPFFT_C2R);

   // hipDeviceSynchronize();

    // Transform signal and kernel
    hipfftExecC2R(planBack, fftSubtomoOut, fft_out);

    float normFactor = volumeSize*volumeSize;

    gpu_normalizedSquare << <numBlocks1D, threadsPerBlock1D >> >(fft_out, normFactor, volumeSize);

   // hipDeviceSynchronize();

    hipfftHandle planForward;
    hipfftPlan3d(&planForward, volumeDim.x, volumeDim.y, volumeDim.z, HIPFFT_R2C);

    hipfftExecR2C(planForward, fft_out, conjugateSquare);

    hipfftDestroy(planForward);
    hipfftDestroy(planBack);
	hipFree(fft_out);
}

void GPURoutines::maskSubtomogram(vector<float>& subtomogram, vector<float>& mask)
{
    // Create a stream for async copy
    //hipStream_t stream1;
   // hipStreamCreate(&stream1);

    //copy mask
    float* dMask;
    gpuErrchk(hipMalloc((void **)&dMask, fftSize*sizeof(float)));
    // copy subtomogram
    float* fftIn;
    gpuErrchk(hipMalloc((void **)&fftIn, volumeSize*sizeof(float)));
    
    
    gpuErrchk(hipMemcpy(dMask, &mask[0], fftSize*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(fftIn, &subtomogram[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));

    // Calculate shifted fourier transform of rotated reference
    hipfftExecR2C(planR2C, fftIn, fftSubtomoOut);

    //hipStreamSynchronize(stream1);
    dim3 fftDim(volumeDim.x, volumeDim.y, fftz);
    complexMultiplicationDDD << <numBlocks1D, threadsPerBlock1D >> > (fftSubtomoOut, dMask, fftDim);

    if (useRosemanCC)
    {
        computeConjugateOfSquare();
        gpu_complexConjugate << <numBlocks1D, threadsPerBlock1D >> > (fftSubtomoOut, conjugateSquare, fftSize);
    }
    else
    {
        double normFactor = computeNormFactor(fftSubtomoOut, fftSize);

        normFactor = volumeSize / sqrt(normFactor);
        // Calculate cross correlation and apply rotated ccmask
        gpu_complexMultiplication << <numBlocks1D, threadsPerBlock1D >> >(fftSubtomoOut, normFactor, fftSize);
    }

    hipFree(fftIn);
    hipFree(dMask);
    
    //hipStreamDestroy(stream1);
}


void GPURoutines::computeCC(vector<float>& ccVolume, vector<float>& shiftedWedgeMask)
{
    
    hipfftComplex* fftRef;
    gpuErrchk(hipMalloc((void **)&fftRef, fftSize*sizeof(hipfftComplex)));

    float* deviceCCVolume;
    gpuErrchk(hipMalloc((void **)&deviceCCVolume, volumeSize*sizeof(float)));

  //  vector<float> rotRef(volumeSize);
  //  gpuErrchk(hipMemcpy(&rotRef[0], deviceRefRot, volumeSize*sizeof(float), hipMemcpyDeviceToHost));
    
    // Calculate shifted fourier transform of rotated reference
    hipfftExecR2C(planR2C, deviceRefRot, fftRef);

    // Allocate device memory for the shifted wedge mask
    float* deviceShiftedWedge;
    gpuErrchk(hipMalloc((void **)&deviceShiftedWedge, fftSize*sizeof(float)));
    gpuErrchk(hipMemcpy(deviceShiftedWedge, &shiftedWedgeMask[0], fftSize*sizeof(float), hipMemcpyHostToDevice));

    // Apply bandpass filtered wedge
    dim3 fftDim(volumeDim.x, volumeDim.y, fftz);
    complexMultiplicationDDD << < numBlocks1D, threadsPerBlock1D >> > (fftRef, deviceShiftedWedge, fftDim);
    
    hipFree(deviceShiftedWedge);

    if (useRosemanCC)
    {

        // Get real space version of masked reference
        hipfftExecC2R(planC2R, fftRef, deviceCCVolume);
        division << <numBlocks1D, threadsPerBlock1D >> > (deviceCCVolume, volumeSize, volumeSize);

        // sum rotated Mask
        double maskSum = sumArray(deviceMaskRot, volumeSize);

        // mask and normalize reference
        float* deviceMaskedRef;
        gpuErrchk(hipMalloc((void **)&deviceMaskedRef, volumeSize*sizeof(float)));
        gpu_pointwiseMult << < numBlocks1D, threadsPerBlock1D >> > (deviceMaskRot, deviceCCVolume, deviceMaskedRef, volumeSize);

        double maskedMean = sumArray(deviceMaskedRef, volumeSize);
        maskedMean = maskedMean / maskSum;

        // Normalization factor of references
        gpu_subtract << < numBlocks1D, threadsPerBlock1D >> >(deviceMaskedRef, maskedMean, volumeSize);

        double sigmaRef = sumArray(deviceMaskedRef, volumeSize, true);
        sigmaRef = sqrt(sigmaRef);

        // Fourier transform of masked ref
        hipfftComplex* maskedFFT;
        gpuErrchk(hipMalloc((void **)&maskedFFT, fftSize*sizeof(hipfftComplex)));
        hipfftExecR2C(planR2C, deviceMaskedRef, maskedFFT);

        hipFree(deviceMaskedRef);

        // Convolution of masked reference and subtomo
        complexMultiplication << < numBlocks1D, threadsPerBlock1D >> >(maskedFFT, fftSubtomoOut, fftSize);

        hipfftExecC2R(planC2R, maskedFFT, numerator);
        division << <numBlocks1D, threadsPerBlock1D >> > (numerator, volumeSize, volumeSize);

        // Fourier transform of mask
        hipfftExecR2C(planR2C, deviceMaskRot, maskedFFT);

        // Mean of subtomo under mask
        gpu_complexMultiplication << < numBlocks1D, threadsPerBlock1D >> >(maskedFFT, fftSubtomoOut, meanMasked, fftSize);

        hipfftExecC2R(planC2R, meanMasked, meanSubtomo);
        division << <numBlocks1D, threadsPerBlock1D >> > (meanSubtomo, volumeSize, volumeSize);

        // Mean intensity of subtomo under mask
        complexMultiplication << < numBlocks1D, threadsPerBlock1D >> >(maskedFFT, conjugateSquare, fftSize);

        hipfftExecC2R(planC2R, maskedFFT, intensitySubtomo);
        division << <numBlocks1D, threadsPerBlock1D >> > (intensitySubtomo, volumeSize, volumeSize);

        // Calculate denominator (of eq 5 in paper)
        gpu_computeFLCC << < numBlocks, threadsPerBlock >> >(deviceCCVolume, numerator, intensitySubtomo, meanSubtomo, maskSum, sigmaRef, volumeDim);
        gpu_multiplication << <numBlocks1D, threadsPerBlock1D >> >(deviceCCVolume, deviceCCMaskRot, volumeSize);
        
        hipFree(maskedFFT);

    }
    else
    {
        double normFactor = computeNormFactor(fftRef, fftSize);
        normFactor = volumeSize / sqrt(normFactor);

        // Calculate cross correlation and apply rotated ccmask
        gpu_conjugateMultiplication << <numBlocks1D, threadsPerBlock1D >> >(fftRef, fftSubtomoOut, normFactor, fftSize);

        // Calculate shifted fourier transform of rotated reference
        hipfftExecC2R(planC2R, fftRef, deviceRefRot);
        division << <numBlocks1D, threadsPerBlock1D >> > (deviceRefRot, volumeSize, volumeSize);

        dim3 halfDim(floor(volumeDim.x / 2.0f), floor(volumeDim.y / 2.0f), floor(volumeDim.z / 2.0f));

        gpu_applyCCMask << <numBlocks, threadsPerBlock >> >(deviceRefRot, deviceCCMaskRot, deviceCCVolume, (float)volumeSize, volumeDim, halfDim);
    }

    gpuErrchk(hipMemcpy(&ccVolume[0], deviceCCVolume, volumeSize*sizeof(float), hipMemcpyDeviceToHost));
    hipFree(deviceCCVolume);
    hipFree(fftRef);

}

void GPURoutines::rotateTexture(Quaternion& rotation)
{
    float4 q;
    q.x = rotation.i;
    q.y = rotation.j;
    q.z = rotation.k;
    q.w = rotation.w;

    gpu_rotateTexture << <numBlocks, threadsPerBlock >> >(deviceRefRot, deviceMaskRot, deviceCCMaskRot, q, volumeDim.x, volumeDim.y, volumeDim.z, rotationType);
    
   // gpu_rotateVolume << <numBlocks, threadsPerBlock >> >(rrrRef, deviceRefRot, rrrMask, deviceMaskRot, rrrCCMask, deviceCCMaskRot, volumeDim.x, volumeDim.y, volumeDim.z, q, rotationType);

}

void GPURoutines::rotateSubtomogramAndWedge(vector<float>& subtomo, vector<float>& wedge, Quaternion& rotation,bool binarizeMask)
{
    float4 q;
    q.x = rotation.i;
    q.y = rotation.j;
    q.z = rotation.k;
    q.w = rotation.w;

    if (!subtomoAllocated)
    {
        hipMalloc((void**)&fftSubtomoInNew, volumeSize*sizeof(float));
        gpuErrchk(hipMemcpy(fftSubtomoInNew, &subtomo[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));
        subtomoAllocated = true;
    }

    copyParamsSubtomo.srcPtr = make_hipPitchedPtr((void *)fftSubtomoInNew, volumeExtent.width*sizeof(float), volumeExtent.width, volumeExtent.height);

    // Copy 3D array from device to device
    gpuErrchk(hipMemcpy3D(&copyParamsSubtomo));

    // Bind array to 3D texture
    gpuErrchk(hipBindTextureToArray(texSubtomo, deviceSubtomo, channelDesc));

   // float* volumeInput;
  //  float* wedgeInput;
  //  float* dummyIn;
  //  float* dummyOut;
  //  gpuErrchk(hipMalloc((void **)&volumeInput, volumeSize*sizeof(float)));
  //  gpuErrchk(hipMemcpy(volumeInput, &subtomo[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));

//    gpuErrchk(hipMalloc((void **)&wedgeInput, volumeSize*sizeof(float)));
  //  gpuErrchk(hipMemcpy(wedgeInput, &wedge[0], volumeSize*sizeof(float), hipMemcpyHostToDevice));

	if (binarizeMask)
		gpu_rotateTexture << <numBlocks, threadsPerBlock >> >(deviceSubtomoRot, deviceWedgeRot, deviceCCMaskRot, q, volumeDim.x, volumeDim.y, volumeDim.z, 1);
	else
		gpu_rotateTexture << <numBlocks, threadsPerBlock >> >(deviceSubtomoRot, deviceWedgeRot, deviceCCMaskRot, q, volumeDim.x, volumeDim.y, volumeDim.z, 4);
 
 //   gpu_rotateVolume << <numBlocks, threadsPerBlock >> >(fftSubtomoInNew, deviceSubtomoRot, wedgeInput, deviceWedgeRot, dummyIn, dummyOut, volumeDim.x, volumeDim.y, volumeDim.z, q, 1);

    gpuErrchk(hipFree(fftSubtomoInNew));
    subtomoAllocated = false;
   // gpuErrchk(hipFree(wedgeInput));

}

void GPURoutines::getRotatedSubtomoAndWedge(vector<float>& subtomo, vector<float>& wedge)
{
    hipMemcpy(&subtomo[0], deviceSubtomoRot, volumeSize*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&wedge[0], deviceWedgeRot, volumeSize*sizeof(float), hipMemcpyDeviceToHost);
}

void GPURoutines::getRotatedReferenceAndCCMask(vector<float>& ref, vector<float>& ccMask)
{
    hipMemcpy(&ref[0], deviceRefRot, volumeSize*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&ccMask[0], deviceCCMaskRot, volumeSize*sizeof(float), hipMemcpyDeviceToHost);
}

void GPURoutines::getFFTResult(vector<float>& result)
{
    hipMemcpy(&result[0], fftSubtomoInNew, volumeSize*sizeof(float), hipMemcpyDeviceToHost);
}

void GPURoutines::getMaskedSubtomogram(vector<float>& outReal, vector<float>& outImag)
{
    vector<float2> output(outReal.size());

    hipMemcpy(&output[0], fftSubtomoOut, outReal.size()*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < outReal.size(); i++)
    {
        outReal[i] = output[i].x;
        outImag[i] = output[i].y;
    }
}

void GPURoutines::getShiftFilter(vector<float>& filterReal, vector<float>& filterImag)
{
    //size_t fftz = floor((float)volumeDim.z / 2) + 1;
   
    vector<float2> filter(volumeDim.x*volumeDim.y*fftz);
    hipMemcpy(&filter[0], shiftFilter, volumeDim.x*volumeDim.y*fftz*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < filterReal.size(); i++)
    {
        filterReal[i] = filter[i].x;
        filterImag[i] = filter[i].y;
    }
}

void GPURoutines::getShiftFilter(vector<float2>& filter)
{
    //size_t fftz = floor((float)volumeDim.z / 2) + 1;

    hipMemcpy(&filter[0], shiftFilter, volumeDim.x*volumeDim.y*fftz*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
}

void GPURoutines::getReferenceAndCCMask(vector<float>& reference, vector<float>& ccmask)
{
    hipMemcpy3DParms copyParamsReference = { 0 };
    copyParamsReference.dstPtr = make_hipPitchedPtr((void *)&reference[0], volumeExtent.width*sizeof(float), volumeExtent.width, volumeExtent.height);
    copyParamsReference.extent = volumeExtent;
    copyParamsReference.kind = hipMemcpyDeviceToHost;
    copyParamsReference.srcArray = deviceRef;

    // Copy 3D array from host to device
    gpuErrchk(hipMemcpy3D(&copyParamsReference));

    copyParamsReference.dstPtr = make_hipPitchedPtr((void *)&ccmask[0], volumeExtent.width*sizeof(float), volumeExtent.width, volumeExtent.height);
    copyParamsReference.srcArray = deviceCCMask;

    // Copy 3D array from host to device
    gpuErrchk(hipMemcpy3D(&copyParamsReference));
}
